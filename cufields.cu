#include "hip/hip_runtime.h"
// This code is dedicated to calculate the magnetostatic field and the demag field and covers the 5th task of Applications of HPC exam
// This code is coming with GNU licence and no warranty  is provided
// This code is accelerated using Cuda platform provided by nVidia 
// (C) Author exam number: Y3833878
// last update: 22nd of April, 2017
// This code is free to use and it can be downloaded from following link: https://github.com/maxxwave/dipole
//
// -----------------------------------------------------------------------------------------------------------------------------------------

#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
// this library is needed for our further fft method which is under progress
#include <hipfft/hipfft.h>
#include <stdio.h>
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/cufields.h"
namespace cuda{
   // defining the pre-factor constant as a register for speed up the access
   //check_cuda_errors (__FILE__, __LINE__);
   // we declare this constant as a register to be very fast each time when it is called

   __device__ __constant__ double cupf =  0.00009274;;	// this is the pre-factor that incorporates the magnetic moment
   // and other. See field.cpp line 51
   __device__ __constant__ double cupf_2 = 8.0*3.14*0.3333333333333;
   __global__ void demag_field(const unsigned long int  index,
				    double * sx_d,
				    double * sy_d,
				    double * sz_d,
                double * Hx_d,
				    double * Hy_d,
				    double * Hz_d,
				    double * x_d,
				    double * y_d,
				    double * z_d,
				    double * H_tot_d ){
       // declaring the radii components as a register in order to speed up the calculations 
       register double dx=0.0;
       register double dy=0.0;
       register double dz=0.0;
       register double r=0.0;
       register double r_cube=0.0;             

      // a synchronization point is needed to ensure all threads will work simultaneously 
      __syncthreads();

      // loop over all threads and assign a thread per atom
      // this loop is dedicated to cover all atoms(sites) if the number of atoms is larger than the maximum number of threads allowed
      for ( register int tdx = blockIdx.x * blockDim.x + threadIdx.x;	//here we define the index
         tdx < index;
         tdx += blockDim.x * gridDim.x // we load a maximum threads allowed
         ){	
         for(int j=0; j < index; j++){
	        if(tdx!=j){
	           dx = x_d[j] - x_d[tdx];
	           dy = y_d[j] - y_d[tdx];
		       dz = z_d[j] - z_d[tdx];
		       r =  sqrt(dx*dx + dy*dy + dz*dz);	
		       r_cube = 1.0/r*r*r;
		       Hx_d[tdx] += cupf * (3.0*dx*(dx*sx_d[j]) - sx_d[j])*r_cube; 	
		       Hy_d[tdx] += cupf * (3.0*dy*(dy*sy_d[j]) - sy_d[j])*r_cube; 	
		       Hz_d[tdx] += cupf * (3.0*dz*(dz*sz_d[j]) - sz_d[j])*r_cube; 	
               
             }// end of if 
             else{
                 Hx_d[tdx] += cupf*(cupf_2*sx_d[tdx]);
                 Hy_d[tdx] += cupf*(cupf_2*sy_d[tdx]);
                 Hz_d[tdx] += cupf*(cupf_2*sz_d[tdx]);
             }// end of else 
               
               // synchronization point 
               __syncthreads();

          } // end of j for	


          H_tot_d[0] += Hx_d[tdx];
	      H_tot_d[1] += Hy_d[tdx];
	      H_tot_d[2] += Hz_d[tdx];

          //synchronization point 
          __syncthreads();

      } //end of tdx for 

   } // end of demag kerne


}//end of namepscare

