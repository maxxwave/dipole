#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipfft/hipfft.h>
#include <stdio.h>
//#include <thrust/device_vector.h>
// include headers
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/cufields.h"
namespace cuda{

	// defining the prefactor constant as a register for speed up the access
	
	 //check_cuda_errors (__FILE__, __LINE__);
	// we declare this constant as a register to be very fast each time when it is called
	__device__ __constant__ double cupf = 1.0e23;	// this is the pre-factor that incorporates the magnetic moment and other constants

	__global__ void demag_field(const unsigned long int  index,
				    double * sx_d,
				    double * sy_d,
				    double * sz_d,
               	    double * Hx_d,
				    double * Hy_d,
				    double * Hz_d,
				    double * x_d,
				    double * y_d,
				    double * z_d,
				    double * H_tot_d )
	{              
	           __syncthreads();
		// loop over all threads and assign a thread per atom
		// this loop is dedicated to cover all atoms(sites) if the number of atoms is larger than the maximum number of threads allowed
		for ( register int tdx = blockIdx.x * blockDim.x + threadIdx.x;	//here we define the index
			tdx < index;
			tdx += blockDim.x * gridDim.x // we load a maximum threads allowed
		){	
			for(int j=0; j < index; j++){
				if(tdx!=j){
	           register double dx=0.0;
               register double dy=0.0;
               register double dz=0.0;
               register double r=0.0;
               register double r_cube=0.0;	
				dx = x_d[j] - x_d[tdx];
				dy = y_d[j] - y_d[tdx];
				dz = z_d[j] - z_d[tdx];
				r =  sqrt(dx*dx + dy*dy + dz*dz);	
				r_cube = 1.0/r*r*r;
				Hx_d[tdx] += cupf * (3.0*dx*(dx*sx_d[j]) - sx_d[j])*r_cube; 	
				Hy_d[tdx] += cupf * (3.0*dy*(dy*sy_d[j]) - sy_d[j])*r_cube; 	
				Hz_d[tdx] += cupf * (3.0*dz*(dz*sz_d[j]) - sz_d[j])*r_cube; 	
	          //  __syncthreads();
                
				}
			} // end of j for	
			H_tot_d[0] += Hx_d[tdx];
			H_tot_d[1] += Hy_d[tdx];
			H_tot_d[2] += Hz_d[tdx];

            __syncthreads();
		
		} //end of tdx for 

		//printf("plm campul de pe placa video e %d \n", &H_tot_d[2]);
		//return EXIT_SUCCESS;
	} // end of demag kernel


}//end of namepscare

