// This code is dedicated to calculate the magnetostatic field and the demag field and covers the 5th task of Applications of HPC exam
// This code is coming with GNU licence and no guaranty is provided
// This code is accelerated using Cuda platform provided by nVidia 
// Author: XXX@york.ac.uk 
// last update: 22nd of April
// This code is free to use and it can be downloaded from following link: https://github.com/maxxwave/dipole
//
// -----------------------------------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
//#include "../hdr/typedef.h"
#include "../hdr/cufields.h"
//#include "../hdr/initialize_gpu.h"

int main(){

   // Initialization part & creating the structure----------------------------------------------------------
   create::create_sc();
   // creating the ellipsoid shape
   create::elli(create::part_origin, create::r_e);
   //
   std::cout << create::index << std::endl;
   // --------------------------------------------------------------------------------------------------------
   //for(int i =0; i<create::index; i++){
   //     std::cout<<st::atom[i].cx<<"\t"<<st::atom[i].cy<<"\t"<<st::atom[i].cz <<"\t"<<st::atom[i].sz<<std::endl;
   //   }

   std::cout<<"Starting to calculate the demag field"<< std::endl;
   std::cout<<"Total number of atoms is:"<<"\t"<<create::index<<std::endl;
   fields_t::demag_fields();
   std::cout<<st::H_total[0]<<"\t"<<st::H_total[1]<<"\t"<<st::H_total[2]<<std::endl;

   //---------------------------------------------------------------------------------------
   // Cuda part
   // --------------------------------------------------------------------------------------
   // #ifdef CUDA
   //initialize GPU 
   std::cout<<"Starting calculate the demag field on CUDA:"<<std::endl; 
   // get number of blocks & threads from dee(GPU)
	// each block contains a certain number of threads
	// a device contains a certain number of blocks depending on the device capabilities 
	const unsigned int block = 1024; // threads per block													
	const unsigned int grid  = 1; // blocks per grid
	

   // declare host arrays corresponding to coordinates, spins values, and fields
   //-----------------------------------------------------------------------------------------------------
   // defining & initializing the pointers corresponding to each host array
    double * d_x=NULL,  * d_y=NULL,  * d_z=NULL;   // coordinates x,y,z
    double * d_Hx=NULL, * d_Hy=NULL, * d_Hz=NULL;  // fields x,y,z
    double * d_sx=NULL, * d_sy=NULL, * d_sz=NULL;  // spin components x,y,z
    double * d_H_total=NULL;                       // total field

   // defining the host vectors (the vectors dedicated to store information on CPU's memory) 
   // coordinates arrays
   std::vector <double> atom_h_x;
   std::vector <double> atom_h_y;
   std::vector <double> atom_h_z;
   // spin arrays
   std::vector <double> atom_h_sx;
   std::vector <double> atom_h_sy;
   std::vector <double> atom_h_sz;
   // fields arrays
   std::vector <double> atom_h_Hx;
   std::vector <double> atom_h_Hy;
   std::vector <double> atom_h_Hz;
   // total field
   std::vector <double> H_total;
   
   // resizing (with the total number of atoms inside the system) & initialization of the host arrays 
   atom_h_x.resize(create::index,0.0);
   atom_h_y.resize(create::index,0.0);
   atom_h_z.resize(create::index,0.0);
   
   atom_h_sx.resize(create::index,0.0);
   atom_h_sy.resize(create::index,0.0);
   atom_h_sz.resize(create::index,0.0);

   atom_h_Hx.resize(create::index,0.0);
   atom_h_Hy.resize(create::index,0.0);
   atom_h_Hz.resize(create::index,0.0);

   H_total.resize(3,0.0);
  
   // within this loop we convert the spin array and coordinates into unidimensional array being more flexible to pass to cuda kernel
   for(int i = 0; i<create::index; i++){ 

      atom_h_x[i]=st::atom[i].cx;
      atom_h_y[i]=st::atom[i].cy;
      atom_h_z[i]=st::atom[i].cz;
      
      atom_h_sx[i]=st::atom[i].sx;
      atom_h_sy[i]=st::atom[i].sy;
      atom_h_sz[i]=st::atom[i].sz;
   }// end of for loop
   
   // this part of the cumain function is dedicated to allocate memory onto device 
   // we use old directives for cuda making sure that the portability is provided for any Cuda version
   // these directives are a bit faster than thrust tools
   // it has been allocated enough memory to achieve a good accuracy through the double precision offered by Cuda
   //----------------------------------------------------------------------------------------------------------
   hipMalloc((void**)&d_x, sizeof(double)*create::index);
   hipMalloc((void**)&d_y, sizeof(double)*create::index);
   hipMalloc((void**)&d_z, sizeof(double)*create::index);

   hipMalloc((void**)&d_sx, sizeof(double)*create::index);
   hipMalloc((void**)&d_sy, sizeof(double)*create::index);
   hipMalloc((void**)&d_sz, sizeof(double)*create::index);

   hipMalloc((void**)&d_Hx, sizeof(double)*create::index);
   hipMalloc((void**)&d_Hy, sizeof(double)*create::index);
   hipMalloc((void**)&d_Hz, sizeof(double)*create::index);
   hipMalloc((void**)&d_H_total, sizeof(double)*3);
  
   // within this step we copy data from host arrays to device array through the device pointers
   hipMemcpy(d_x, &atom_h_x[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_y, &atom_h_y[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_z, &atom_h_z[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 

   hipMemcpy(d_sx, &atom_h_sx[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_sy, &atom_h_sy[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_sz, &atom_h_sz[0], sizeof(double)*create::index, hipMemcpyHostToDevice);

   hipMemcpy(d_Hx, &atom_h_Hx[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_Hy, &atom_h_Hy[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_Hz, &atom_h_Hz[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_H_total, &H_total[0], sizeof(double)*3, hipMemcpyHostToDevice); 

   // it is important to delete the memory while operations are performed on device in order to optimize the memory 
   atom_h_x.clear();
   atom_h_y.clear();
   atom_h_z.clear();
   atom_h_sx.clear();
   atom_h_sy.clear();
   atom_h_sz.clear();
   atom_h_Hx.clear();
   atom_h_Hy.clear();
   atom_h_Hz.clear();
   H_total.clear();

   // synchronization of device in order to ensure that all threads works simultaneously 
   hipDeviceSynchronize();

   // we call kernel function and execution--------------------------------------------------------------
   cuda::demag_field <<< grid,block >>> (create::index,
					   d_sx,
					   d_sy,
					   d_sz,
					   d_Hx,
					   d_Hy,
					   d_Hz,
					   d_x,
					   d_y,
					   d_z,
					   d_H_total
					  ); 
   //------------------------------------------------------------------------------------------------------
   hipDeviceSynchronize();
   // copy data from device to host and de-allocate the memory into device  
   hipMemcpy(&atom_h_x[0], d_x, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_y[0], d_y, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_z[0], d_z, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_sx[0], d_sx, sizeof(double)*create::index, hipMemcpyDeviceToHost);  
   hipMemcpy(&atom_h_sy[0], d_sy, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_sz[0], d_sz, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hx[0], d_Hx, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hy[0], d_Hy, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hz[0], d_Hz, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&H_total[0], d_H_total, sizeof(double)*3, hipMemcpyDeviceToHost); 
   hipDeviceSynchronize();
   // dellocate memory from device
	hipFree(d_x); 
	hipFree(d_y); 
	hipFree(d_z); 
	hipFree(d_sx); 
	hipFree(d_sy); 
	hipFree(d_sz); 
	hipFree(d_Hx); 
	hipFree(d_Hy); 
	hipFree(d_Hz); 
	hipFree(d_H_total); 
	// print out results

	for(int i =0; i<create::index; i++){
	   std::cout<<atom_h_x[i]<<"\t"
		<<atom_h_y[i]<<"\t"
		<<atom_h_z[i]<<"\t"
		<<atom_h_sx[i]<<"\t"
		<<atom_h_sy[i]<<"\t"
		<<atom_h_sz[i]<<"\t"
		<<atom_h_Hx[i]<<"\t"
		<<atom_h_Hy[i]<<"\t"
		<<atom_h_Hz[i]<<"\t"
		<<std::endl;
      } 

    std::cout<<H_total[0]<<"\t"<<H_total[1]<<"\t"<<H_total[2]<<std::endl;
   
   //#endif

   // print the results part!
   // TODO
   return 0;
}
