#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipfft/hipfft.h>

// include headers
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/cufields.h"
#include "../hdr/typedef.h"


namespace cuda{
	// defining the prefactor constant as a register for speed up the access
	//__constant__ cu_real_t pf = 10e23;

	__global__ void demag_field(long int  index,
				    double * sx_d,
				    double * sy_d,
				    double * sz_d,
				    double * Hx_d,
				    double * Hy_d,
				    double * Hz_d,
				    double * x_d,
				    double * y_x,
				    double * z_d,
				    double * H_tot_d )
	{
		
		//defining the index
		int tdx = blockIdx.x * blockDim.x + threadIdx.x;
		

		//return EXIT_SUCCESS;
	} // end of demag kernel


}//end of namepscare

