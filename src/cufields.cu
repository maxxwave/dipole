#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipfft/hipfft.h>

// include headers
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/cufields.h"
#include "../hdr/typedef.h"
#include "../hdr/initialize_gpu.h"

namespace cuda{
	// defining the prefactor constant as a register for speed up the access
	
	 //check_cuda_errors (__FILE__, __LINE__);
	// we declare this constat as a register to be very fast each time when it is called
	__device__ __constant__ double cupf = 10e23;	// this is the prefactor that incorporates the magnetic moment and other constants

	__global__ void demag_field(long int  index,
				    cu_real_t * sx_d,
				    cu_real_t * sy_d,
				    cu_real_t * sz_d,
				    cu_real_t * Hx_d,
				    cu_real_t * Hy_d,
				    cu_real_t * Hz_d,
				    cu_real_t * x_d,
				    cu_real_t * y_x,
				    cu_real_t * z_d,
				    cu_real_t * H_tot_d )
	{
		// loop over all threads and assign a thread per atom
		// this loop is dedicated to cover all atoms(sites) if the number of atoms is larger than the maximum number of threads allowed
		for ( int tdx = blockIdx.x * blockDim.x + threadIdx.x;	//here we define the index
			tdx < index;
			tdx += blockDim.x * gridDim.x // we load a maximum threads allowed
		){

			for(int j=0; j < index; j++){
				
				cu_real_t dx = sx_d[j] - sx_d[tdx];
				cu_real_t dy = sy_d[j] - sy_d[tdx];
				cu_real_t dz = sx_d[j] - sz_d[tdx];
				
				cu_real_t r = sqrt(dx*dx + dy*dy + dz*dz);	
				cu_real_t r_cube = 1.0/r*r*r;

				Hx_d[tdx] += cupf * (3.0*dx*(dx*sx_d[j]) - sx_d[j])*r_cube; 	
				Hy_d[tdx] += cupf * (3.0*dy*(dy*sy_d[j]) - sy_d[j])*r_cube; 	
				Hz_d[tdx] += cupf * (3.0*dz*(dz*sz_d[j]) - sz_d[j])*r_cube; 	
	
		
			} // end of j for	
			H_tot_d[0] += Hx_d[tdx];
			H_tot_d[1] += Hy_d[tdx];
			H_tot_d[2] += Hz_d[tdx];
		} //end of tdx for 
		//return EXIT_SUCCESS;
	} // end of demag kernel


}//end of namepscare

