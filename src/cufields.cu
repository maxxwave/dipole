#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipfft/hipfft.h>

// include headers
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/cufields.h"

namespace cuda{
	// defining the prefactor constant as a register for speed up the access
	//__constant__ float pf = 10e23;

	__global__ void demag_field(int index,
				    cu_real_array_t * sx_d,
				    cu_real_array_t * sy_d)
	{

		//defining the index
		int tdx = blockIdx.x * blockDim.x + threadIdx.x;

		//return EXIT_SUCCESS;
	} // end of demag kernel


}//end of namepscare
