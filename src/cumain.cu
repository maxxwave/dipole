#include "hip/hip_runtime.h"
// This code is dedicated to calculate the magnetostatic field and the demag field and covers the 5th task of Applications of HPC exam
// This code is coming with GNU licence and no warranty  is provided
// This code is accelerated using Cuda platform provided by nVidia 
// (C) Author: exam no Y3833878
// last update: 22nd of April, 2017
// This code is free to use and it can be downloaded from following link: https://github.com/maxxwave/dipole
//
// -----------------------------------------------------------------------------------------------------------------------------------------


#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <time.h>
#include <cstdlib>
//include demag headers
#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
#include "../hdr/cufields.h"
#include "../hdr/demag.h"


int main(int argc, char* argv[]){
   //-----------------------------------------------------------------------------------------------------------
   // 1.0: Initialization part & creating the structure
   //-----------------------------------------------------------------------------------------------------------
   create::create_sc();
   
   //-----------------------------------------------------------------------------------------------------------
   // 2.0: Creating the ellipsoid shape
   //-----------------------------------------------------------------------------------------------------------

   // counting the time elapsed for demag routine on CPU
   // declaring the start and stop times
   clock_t start_1, stop_1;

   // calling the structure making routine
   create::elli(create::part_origin, create::r_e);

   // printing an informative message with the total number of particles including the particles outside of the ellipsoid
   std::cout <<"The total number of particles:"<<"\t"<<create::index << std::endl;
   std::cout <<"\n"<<std::endl;
   std::cout <<"Starting to calculate the demag field"<< std::endl;

   //click on start 
   start_1 = clock();
   // calling the field routine on CPU
   fields_t::demag_fields();
   // click on stop
   stop_1 = clock();

   //printing the time elapsed
   std::cout<<"Time on CPU"<<"\t"<<(stop_1-start_1)/CLOCKS_PER_SEC<<std::endl;

   //-------------------------------------------------------------------------------------------------------------
   // 2.1: printing the results
   //-------------------------------------------------------------------------------------------------------------
   // open file for output
   std::ofstream outfile;
   outfile.open("results_on_CPU.data");

   // printing the time elapsed for performing calculation on CPU
   outfile<<"Time on CPU"<<"\t"<<(stop_1-start_1)/CLOCKS_PER_SEC<<std::endl;

   //loop over all sites and print the field values
   for (int i=0; i<create::index; i++){

      // printing the fields values for each particle
      outfile<<st::H_dip[i].x<<"\t"<<st::H_dip[i].y<<"\t"<<st::H_dip[i].z<<std::endl;
      }// end of for

   // close the file
   outfile.close();
   
   //------------------------------------------------------------------------------------------------------------
   // 3.0: CUDA part
   // -----------------------------------------------------------------------------------------------------------
    
   //initialize GPU 
   std::cout<<"Starting calculate the demag field on CUDA:"<<std::endl; 
   // get number of blocks & threads from dee(GPU)
   // each block contains a certain number of threads
   // a device contains a certain number of blocks depending on the device capabilities 
   int block = atoi(argv[1]); // threads per block													
   int grid  = atoi(argv[2]); // blocks per grid
   
   // check right input parameters for getting number of threads and blocks
   if((block !=0) && (grid!=0) && (block<=1024)) { 
      //print an informative message
      std::cout<<"your cuda program has been successfully initialized with:" <<"\t"<<block<<"\t"<<"threads"<<"\t"<<"and"<<"\t"<<grid<<"\t"<<"blocks"<<std::endl;
   }// end of if
   else { //print an error msg
       std::cout<<"Please make sure your parameter are appropriate number for number of threads and blocks!"<<std::endl;
       std::cout<<"Be aware on your gpu card architecture. On Fermi architecture you are allowed to use only 1024 threads per block!"<<std::endl;
   } 

   // declare host arrays corresponding to coordinates, spins values, and fields
   // defining & initializing the pointers corresponding to each host array
   double * d_x=NULL,  * d_y=NULL,  * d_z=NULL;   // coordinates x,y,z
   double * d_Hx=NULL, * d_Hy=NULL, * d_Hz=NULL;  // fields x,y,z
   double * d_sx=NULL, * d_sy=NULL, * d_sz=NULL;  // spin components x,y,z
   double * d_H_total=NULL;                       // total field

   // defining the host vectors (the vectors dedicated to store information on CPU's memory) 
   // coordinates arrays
   std::vector <double> atom_h_x;
   std::vector <double> atom_h_y;
   std::vector <double> atom_h_z;
   // spin arrays
   std::vector <double> atom_h_sx;
   std::vector <double> atom_h_sy;
   std::vector <double> atom_h_sz;
   // fields arrays
   std::vector <double> atom_h_Hx;
   std::vector <double> atom_h_Hy;
   std::vector <double> atom_h_Hz;
   // total field
   std::vector <double> H_total;
   
   // resizing (with the total number of atoms inside the system) & initialization of the host arrays 
   atom_h_x.resize(create::index,0.0);
   atom_h_y.resize(create::index,0.0);
   atom_h_z.resize(create::index,0.0);
   
   atom_h_sx.resize(create::index,0.0);
   atom_h_sy.resize(create::index,0.0);
   atom_h_sz.resize(create::index,0.0);

   atom_h_Hx.resize(create::index,0.0);
   atom_h_Hy.resize(create::index,0.0);
   atom_h_Hz.resize(create::index,0.0);

   H_total.resize(3,0.0);
  
   // within this loop we convert the spin array and coordinates into unidimensional array being more flexible to pass to cuda kernel
   for(int i = 0; i<create::index; i++){ 

      atom_h_x[i]=st::atom[i].cx;
      atom_h_y[i]=st::atom[i].cy;
      atom_h_z[i]=st::atom[i].cz;
      
      atom_h_sx[i]=st::atom[i].sx;
      atom_h_sy[i]=st::atom[i].sy;
      atom_h_sz[i]=st::atom[i].sz;
   }// end of for loop

   // this part of the cumain function is dedicated to allocate memory onto device 
   // we use old directives for cuda making sure that the portability is provided for any Cuda version
   // these directives are a bit faster than thrust tools
   // it has been allocated enough memory to achieve a good accuracy through the double precision offered by Cuda
   // allocate memory for coordinates
   hipMalloc((void**)&d_x, sizeof(double)*create::index);
   hipMalloc((void**)&d_y, sizeof(double)*create::index);
   hipMalloc((void**)&d_z, sizeof(double)*create::index);
   // allocate memory for spin values
   hipMalloc((void**)&d_sx, sizeof(double)*create::index);
   hipMalloc((void**)&d_sy, sizeof(double)*create::index);
   hipMalloc((void**)&d_sz, sizeof(double)*create::index);
   // allocate memory for field values
   hipMalloc((void**)&d_Hx, sizeof(double)*create::index);
   hipMalloc((void**)&d_Hy, sizeof(double)*create::index);
   hipMalloc((void**)&d_Hz, sizeof(double)*create::index);
   hipMalloc((void**)&d_H_total, sizeof(double)*3);
   // within this step we copy data from host arrays to device array through the device pointers
   hipMemcpy(d_x, &atom_h_x[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_y, &atom_h_y[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_z, &atom_h_z[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   // copy spins to device
   hipMemcpy(d_sx, &atom_h_sx[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_sy, &atom_h_sy[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_sz, &atom_h_sz[0], sizeof(double)*create::index, hipMemcpyHostToDevice);
   // copy fields to device
   hipMemcpy(d_Hx, &atom_h_Hx[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_Hy, &atom_h_Hy[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_Hz, &atom_h_Hz[0], sizeof(double)*create::index, hipMemcpyHostToDevice); 
   hipMemcpy(d_H_total, &H_total[0], sizeof(double)*3, hipMemcpyHostToDevice); 

   // it is important to delete the memory while operations are performed on device in order to optimize the memory 
   atom_h_x.clear();
   atom_h_y.clear();
   atom_h_z.clear();
   atom_h_sx.clear();
   atom_h_sy.clear();
   atom_h_sz.clear();
   atom_h_Hx.clear();
   atom_h_Hy.clear();
   atom_h_Hz.clear();
   H_total.clear();
   
   // defining times for measuring the cuda routine
   hipEvent_t start_2, stop_2;
   //clicking on start
   hipEventCreate(&start_2);
   hipEventCreate(&stop_2);
   hipEventRecord(start_2);

   // synchronization of device in order to ensure that all threads works simultaneously 
   hipDeviceSynchronize();
    
   // we call kernel function and execute it 
   cuda::demag_field <<< grid,block >>> (create::index,
					   d_sx,
					   d_sy,
					   d_sz,
					   d_Hx,
					   d_Hy,
					   d_Hz,
					   d_x,
					   d_y,
					   d_z,
					   d_H_total
					  ); 
   // synchronization point 
   hipDeviceSynchronize();
   // we measure only the time elapsed for execution of kernel function
   hipEventRecord(stop_2);
   hipEventSynchronize(stop_2);
   // defining the elapsed time for measuring the bandwidth
   float elapsed_time1 = 0;
   //this will be in milliseconds 

   hipEventElapsedTime(&elapsed_time1, start_2, stop_2);
   // printing the time elapsed for Cuda kernel
   std::cout<<"Cuda time:" <<"\t"<<elapsed_time1/1000<<"s"<<std::endl;
   
   // we can obtain information of the bandwidth of the device by measuring the time elapsed for copying a certain
   // amount of data
   // defining Cuda times
   hipEvent_t start_3, stop_3;
   // create events
   hipEventCreate(&start_3);
   hipEventCreate(&stop_3);
   // start measuring
   hipEventRecord(start_3);

   // copy data from device to host and de-allocate the memory into device  
   hipMemcpy(&atom_h_x[0], d_x, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_y[0], d_y, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_z[0], d_z, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_sx[0], d_sx, sizeof(double)*create::index, hipMemcpyDeviceToHost);  
   hipMemcpy(&atom_h_sy[0], d_sy, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_sz[0], d_sz, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hx[0], d_Hx, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hy[0], d_Hy, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&atom_h_Hz[0], d_Hz, sizeof(double)*create::index, hipMemcpyDeviceToHost); 
   hipMemcpy(&H_total[0], d_H_total, sizeof(double)*3, hipMemcpyDeviceToHost); 
   
   // a synchronization point is needed 
   hipDeviceSynchronize();
   
   // stop the clock
   hipEventRecord(stop_3);
   hipEventSynchronize(stop_3);
   
   float elapsed_time2 =0;
   //computing elapsed time
   hipEventElapsedTime(&elapsed_time2, start_3, stop_3);
   
   // print the bandwidth of the device according to CUDA documentation
   // more information can be found on following website: https://devblogs.nvidia.com/parallelforall/author/mharris/

   std::cout<<"The bandwidth is:"<<"\t"<<
   (sizeof(double)*(9*create::index+3))/(elapsed_time2*1e6)<<"\t"<<"GB/s"<<std::endl;

   // deallocate memory from device: coordinate & spins values & field values & total field
   hipFree(d_x); 
   hipFree(d_y); 
   hipFree(d_z); 
   hipFree(d_sx); 
   hipFree(d_sy); 
   hipFree(d_sz); 
   hipFree(d_Hx); 
   hipFree(d_Hy); 
   hipFree(d_Hz); 
   hipFree(d_H_total); 
	
   // print out results
   std::ofstream cuoutfile;
   // declaring a separate outputfile for cuda results
   cuoutfile.open("Curesults.data");
   // print the header of results file
   cuoutfile << "x"<<"\t"<<"y"<<"\t"<<"z"<<"\t"<<"Hz"<<std::endl;
   // loop over all sites and print the spin and coordinates values
   for (int i=0; i<create::index; i++){
       cuoutfile << atom_h_x[i]<<"\t"
                 << atom_h_y[i]<<"\t"
                 << atom_h_z[i]<<"\t"
                 << atom_h_Hz[i]<<std::endl;
   }// end of for loop
       
   // close the file
   cuoutfile.close();
   
   return 0;
}
