// This program is a part of Dipolar field calculation
//
//
// This routine is dedicated to create the spin, coordinates and field's array required to pass information to GPU
//
// We used CUSP library in order to have double precision 

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>


// these are the libraries needed to manage the array for device

// Dipolar headers
#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
#include "../hdr/typedef.h"


namespace cuda{
	
	void initialize_gpu(){
		
		// get number of blocks & threads from dee(GPU)
		// each block contains a certain number of threads
		// a device contains a certain number of blocks depending on the device capabilities 
		int block = 1024; // 1024 blocks pe grid
													
		int grid  = 128; // 128 grids per device
		///////////////////  1024 X 128 = 131072 threads executed in parallel////////////////
		
		// defining some uni dimensional array
		// in following lines of code we convert the class of spin array and field array into unidimensional array to be able to pass them to kernel
		// coordinates arrays
		std::vector <double> x_coord(create::index);
		std::vector <double> y_coord(create::index);
		std::vector <double> z_coord(create::index);
		// spin arrays
		std::vector <double> sx(create::index);
		std::vector <double> sy(create::index);
		std::vector <double> sz(create::index);
		//demag field arrays	
		std::vector <double> Hx_dip(create::index);
		std::vector <double> Hy_dip(create::index);
		std::vector <double> Hz_dip(create::index);
		// defining a 3-component vector in order to store the summation of all atoms field		 
		std::vector <double> H_total(3,0.0);
	
		// copy the values of the spins and coordinates of the atoms after creating the structure & initialization to new arrays
		for(int i=0; i<create::index ; i++){
			//copying the coordinates
			x_coord[i]=st::atom[i].cx;
			y_coord[i]=st::atom[i].cy;
			z_coord[i]=st::atom[i].cz;
			// copyinh the spins values
			sx[i]=st::atom[i].sx;
			sy[i]=st::atom[i].sy;
			sz[i]=st::atom[i].sz;
			// TODO
			// also for a general case we should create some arrays for the magnitude of the magnetic moment of each atom
		}// end of for loop
		
		// following we create device array and also alocate memory to device
		// a nice way to do it using CUSP library in order to avoid cuMemcpy and other ugly stuff
		// another benefit of this is double precision offered by this typedef (../hdr/typede.h)
		thrust::device_vector<double> sx_d(create::index);
		thrust::device_vector<double> sy_d(create::index);
		thrust::device_vector<double> sz_d(create::index);
		
		// coordinate arrays
		thrust::device_vector<double> x_d(create::index);
		thrust::device_vector<double> y_d(create::index);
		thrust::device_vector<double> z_d(create::index);

		// field arrays
		thrust::device_vector<double> Hx_d(create::index);
		thrust::device_vector<double> Hy_d(create::index);
		thrust::device_vector<double> Hz_d(create::index);
		// total field obtained by summation of each atom	
		thrust::device_vector<double> H_tot_d(create::index);

		// copy data to device 
		// copy spin values
		thrust::copy( sx.begin(), sx.end(), sx_d.begin());
		thrust::copy( sy.begin(), sy.end(), sy_d.begin());
		thrust::copy( sz.begin(), sz.end(), sz_d.begin());
		
		// copy coordinates
		thrust::copy( x_coord.begin(), x_coord.end(), x_d.begin());
		thrust::copy( y_coord.begin(), y_coord.end(), y_d.begin());
		thrust::copy( z_coord.begin(), z_coord.end(), z_d.begin());

		// copy field arrays
		thrust::copy( Hx_dip.begin(), Hx_dip.end(), Hx_d.begin());
		thrust::copy( Hy_dip.begin(), Hy_dip.end(), Hy_d.begin());
		thrust::copy( Hz_dip.begin(), Hz_dip.end(), Hz_d.begin());

		thrust::copy( H_total.begin(), H_total.end(), H_tot_d.begin());

	}// end of routine
}// end of namespace
