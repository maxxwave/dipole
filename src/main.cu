#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
//#include <cusp/copy.h>
//#include <cusp/csr_matrix.h>
//#include <cusp/dia_matrix.h>
//#include <cusp/ell_matrix.h>
//#include <cusp/multiply.h>


#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
<<<<<<< HEAD
#include "../hdr/typedef.h"
=======
>>>>>>> 6551780858f46e4d66a2861ab68cb5fed9b41527
//#include "../hdr/cufields.h"
int main(){

   // run create
   create::create_sc();
   create::elli(create::part_origin, create::r_e);
   std::cout << create::index << std::endl;
   for(int i =0; i<create::index; i++){
        std::cout<<st::atom[i].cx<<"\t"<<st::atom[i].cy<<"\t"<<st::atom[i].cz <<"\t"<<st::atom[i].sz<<std::endl;
      }
   std::cout<<"Starting to calculate the demag field"<< std::endl;
   std::cout<<"Total number of atoms is:"<<"\t"<<create::index<<std::endl;
  // fields_t::demag_fields();
   //std::cout<<st::H_total[0]<<"T"<<std::endl;



   //---------------------------------------------------------------------------------------
   // Cuda part
   // --------------------------------------------------------------------------------------
  // #ifdef CUDA

   //int N=1024;
   int block=128;
   int grid=1024;
   std::vector <double> x_coord;
   x_coord.resize(create::index);
   std::vector <double> y_coord;
   y_coord.resize(create::index);
   std::vector <double> z_coord;
   z_coord.resize(create::index);
   std::vector <double> H_dip_x;
   H_dip_x.resize(create::index);
   std::vector <double> H_dip_y;
   H_dip_y.resize(create::index);
   std::vector <double> H_dip_z;
   H_dip_z.resize(create::index);
   std::vector <double> sx;
   sx.resize(create::index);
   std::vector <double> sy;
   sy.resize(create::index);
   std::vector <double> sz;
   sz.resize(create::index);
   for (int i=0; i<create::index; i++){
       x_coord[i] = st::atom[i].cx;
       y_coord[i] = st::atom[i].cy;
       z_coord[i] = st::atom[i].cz;
       sx[i]      = st::atom[i].sx;
       sy[i]      = st::atom[i].sx;
       sz[i]      = st::atom[i].sx;
   }

<<<<<<< HEAD
  thrust::device_vector<double> H_d(create::index);
  thrust::device_vector<double> sx_d(create::index);
  thrust::device_vector<double> sy_d(create::index);
  thrust::device_vector<double> sz_d(create::index);
 // cu_real_array_t sx_d(0UL);
 // cu_real_array_t sy_d(0UL);
 // cu_real_array_t sx_d(0UL);
=======
//   thrust::device_vector<double> H_d(create::index);
 //  thrust::device_vector<double> sx_d(create::index);
  // thrust::device_vector<double> sy_d(create::index);
  // thrust::device_vector<double> sz_d(create::index);
  cu_real_array_t sx_d(0UL);
  cu_real_array_t sy_d(0UL);
  cu_real_array_t sx_d(0UL);
>>>>>>> 6551780858f46e4d66a2861ab68cb5fed9b41527

   thrust::copy( sx.begin(),
                 sx.end(),
                 sx_d.begin()
                 );

   thrust::copy( sy.begin(),
                 sy.end(),
                 sy_d.begin()
                 );


   //H_d = thrust::raw_pointer_cast(st::H_dip.data());
   // * d_H_dip = thrust::raw_pointer_cast(st::H_dip.data());
   //int size = sizeof(double);
   //hipMalloc((void**)&atom_d, size*3*size);
  // hipMalloc((void**)&H_dip_d, size*3*size);
  // hipMemcpy(atom_d, st::atom, 3*create::index*size, hipMemcpyHostToDevice);
   //hipMemcpy(H_dip_d, st::H_dip, 3*create::index*size, hipMemcpyHostToDevice);
  // cuda::demag_field <<< grid,block >>> (create::index,sx_d,sy_d );

   //#endif
   return 0;
}
<<<<<<< HEAD
=======

>>>>>>> 6551780858f46e4d66a2861ab68cb5fed9b41527
