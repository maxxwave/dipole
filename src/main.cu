#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <cusp/copy.h>
#include <cusp/csr_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/multiply.h>


#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
#include "../hdr/typedef.h"
#include "../hdr/cufields.h"
#include "../hdr/initialize_gpu.h"

int main(){

   // Iinitialization part & creating the structure----------------------------------------------------------
   create::create_sc();
   // creating the ellipsoid shape
   create::elli(create::part_origin, create::r_e);
   //
   std::cout << create::index << std::endl;
   // --------------------------------------------------------------------------------------------------------
/*   for(int i =0; i<create::index; i++){
        std::cout<<st::atom[i].cx<<"\t"<<st::atom[i].cy<<"\t"<<st::atom[i].cz <<"\t"<<st::atom[i].sz<<std::endl;
      }

   std::cout<<"Starting to calculate the demag field"<< std::endl;
   std::cout<<"Total number of atoms is:"<<"\t"<<create::index<<std::endl;
  // fields_t::demag_fields();
   //std::cout<<st::H_total[0]<<"T"<<std::endl;

*/

   //---------------------------------------------------------------------------------------
   // Cuda part
   // --------------------------------------------------------------------------------------
   // #ifdef CUDA
   //initialize GPU 
   cuda::initialize_gpu();

   // in this step we create some associated pointers to our arrays on device to be able to pass into kernel
   //------------------------------------------------------------------------------------------------------
   cuda::cu_real_t * d_sx = thrust::raw_pointer_cast(cuda::sx_d.data());
   cuda::cu_real_t * d_sy = thrust::raw_pointer_cast(cuda::sy_d.data());
   cuda::cu_real_t * d_sz = thrust::raw_pointer_cast(cuda::sz_d.data());
 
   cuda::cu_real_t * d_x = thrust::raw_pointer_cast(cuda::x_d.data());
   cuda::cu_real_t * d_y = thrust::raw_pointer_cast(cuda::y_d.data());
   cuda::cu_real_t * d_z = thrust::raw_pointer_cast(cuda::z_d.data());

   cuda::cu_real_t * d_Hx = thrust::raw_pointer_cast(cuda::Hx_d.data());
   cuda::cu_real_t * d_Hy = thrust::raw_pointer_cast(cuda::Hy_d.data());
   cuda::cu_real_t * d_Hz = thrust::raw_pointer_cast(cuda::Hz_d.data());
   cuda::cu_real_t * d_H_total = thrust::raw_pointer_cast(cuda::H_tot_d.data());
   //-----------------------------------------------------------------------------------------------------
  
   // we call kernel function and execution--------------------------------------------------------------
   cuda::demag_field<<< cuda::block,cuda::grid >>>(create::index,
					   d_sx,
					   d_sy,
					   d_sz,
					   d_Hx,
					   d_Hy,
					   d_Hz,
					   d_x,
					   d_y,
					   d_z,
					   d_H_total
					  ); 
   //------------------------------------------------------------------------------------------------------
  
   // copy data from device to host and de-allocate the memory into device 
   
   
   //#endif

   // print the results part!
   // TODO
   return 0;
}
