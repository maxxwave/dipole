#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <cusp/copy.h>
#include <cusp/csr_matrix.h>
#include <cusp/dia_matrix.h>
#include <cusp/ell_matrix.h>
#include <cusp/multiply.h>


#include "../hdr/template.h"
#include "../hdr/storage.h"
#include "../hdr/create.h"
#include "../hdr/fields.h"
#include "../hdr/typedef.h"
#include "../hdr/cufields.h"
#include "../hdr/initialize_gpu.h"

int main(){

   // run create
   create::create_sc();
   create::elli(create::part_origin, create::r_e);
   std::cout << create::index << std::endl;
   for(int i =0; i<create::index; i++){
        std::cout<<st::atom[i].cx<<"\t"<<st::atom[i].cy<<"\t"<<st::atom[i].cz <<"\t"<<st::atom[i].sz<<std::endl;
      }

   std::cout<<"Starting to calculate the demag field"<< std::endl;
   std::cout<<"Total number of atoms is:"<<"\t"<<create::index<<std::endl;
  // fields_t::demag_fields();
   //std::cout<<st::H_total[0]<<"T"<<std::endl;



   //---------------------------------------------------------------------------------------
   // Cuda part
   // --------------------------------------------------------------------------------------
   // #ifdef CUDA
   //initialize GPU 
   cuda::initialize_gpu(); 
   cuda::demag_field<<< cuda::block,cuda::grid >>>(create::index,
					   cuda::sx_d,
					   cuda::sy_d,
					   cuda::sz_d,
					   cuda::Hx_d,
					   cuda::Hy_d,
					   cuda::Hz_d,
					   cuda::x_d,
					   cuda::y_d,
					   cuda::z_d,
					   cuda::H_tot_d
					  ); 
   //#endif
   return 0;
}
